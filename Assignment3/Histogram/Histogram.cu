#include "hip/hip_runtime.h"
#include <wb.h>
#include <cmath>
#include <stdio.h>
#include <string.h>
#include "hip/hip_runtime.h"
#include ""


#define NUM_BINS 4096
#define SATURATION 127
#define THREADS 256
#define BLOCKS 32

#define CUDA_CHECK(ans)                                                   \
  { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code),
            file, line);
    if (abort)
      exit(code);
  }
}

//@@Add the Kernel Code here
/// Will construct the bins
__global__ void histogram(unsigned int *input, unsigned int *bins, unsigned int inputLength)
{
	
	// BlockDim is inputlength / threads for each block. 
	__shared__ unsigned int bins_s[NUM_BINS];

	unsigned int thread = (blockDim.x * blockIdx.x) + threadIdx.x; // gets the thread Id. 
	unsigned int stride = gridDim.x * blockDim.x; // get the stride. 

	for (int j = threadIdx.x; j < NUM_BINS; j += THREADS)
	{
		if (j < NUM_BINS) // This should make sure we do not excede the BINS
		{
			bins_s[j] = 0;
		}
	}
	__syncthreads();

	unsigned int threadInc = thread; // Sets the thread increment value. 

	while(threadInc < inputLength) // after each increment, check if we are in bounds.
	{
		atomicAdd(&(bins_s[input[threadInc]]), 1); // increment the index in bins

		threadInc += stride; // increment the threadInc
	}

	__syncthreads();
	for (int j = threadIdx.x; j < NUM_BINS; j += THREADS)
	{
		if (j < NUM_BINS) // This should make sure we do not excede the BINS
		{
			atomicAdd(&(bins[j]), bins_s[j]);
		}
	}
	__syncthreads();
	
	for (int j = threadIdx.x; j < NUM_BINS; j += THREADS)
	{
		
		if (bins[j] > SATURATION)
		{
			bins[j] = SATURATION;
		}
	}

	__syncthreads();
}

int main(int argc, char *argv[]) {
  wbArg_t args;
  int inputLength;
  unsigned int *hostInput;
  unsigned int *hostBins;
  unsigned int *deviceInput;
  unsigned int *deviceBins; // output array

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");

  hostInput = (unsigned int *)wbImport(wbArg_getInputFile(args, 0),
                                       &inputLength, "Integer");
  hostBins = (unsigned int *)malloc(NUM_BINS * sizeof(unsigned int));

  wbTime_stop(Generic, "Importing data and creating memory on host");
  wbLog(TRACE, "The input length is ", inputLength); 
  wbLog(TRACE, "The number of bins is ", NUM_BINS);

  wbTime_start(GPU, "Allocating GPU memory.");
  //@@ Allocate GPU memory here
  hipMalloc((void**)&deviceInput, inputLength*sizeof(unsigned int));
  hipMalloc((void**)&deviceBins, sizeof(unsigned int) * NUM_BINS);

  CUDA_CHECK(hipDeviceSynchronize());
  wbTime_stop(GPU, "Allocating GPU memory.");

  wbTime_start(GPU, "Copying input memory to the GPU.");
  //@@ Copy memory to the GPU here
  hipMemcpy(deviceInput, hostInput, inputLength*sizeof(unsigned int), hipMemcpyHostToDevice);
  hipMemset(deviceBins, 0, sizeof(unsigned int)*NUM_BINS);

  CUDA_CHECK(hipDeviceSynchronize());
  wbTime_stop(GPU, "Copying input memory to the GPU.");

  // Launch kernel
  // ----------------------------------------------------------
  wbLog(TRACE, "Launching kernel");

  dim3 gridDim(BLOCKS);
  dim3 blockDim(THREADS);

  wbTime_start(Compute, "Performing CUDA computation");
  //@@ Perform kernel computation here
  histogram << <gridDim, blockDim >> > (deviceInput, deviceBins, inputLength);


  CUDA_CHECK(hipDeviceSynchronize());
  wbTime_stop(Compute, "Performing CUDA computation");

  wbTime_start(Copy, "Copying output memory to the CPU");
  //@@ Copy the GPU memory back to the CPU here
  hipMemcpy(hostBins, deviceBins, sizeof(unsigned int)*NUM_BINS, hipMemcpyDeviceToHost);

  CUDA_CHECK(hipDeviceSynchronize());
  wbTime_stop(Copy, "Copying output memory to the CPU");

  wbTime_start(GPU, "Freeing GPU Memory");
  //@@ Free the GPU memory here

  hipFree(deviceBins);
  hipFree(deviceInput);
  wbTime_stop(GPU, "Freeing GPU Memory");

  // Verify correctness
  // -----------------------------------------------------
  wbSolution(args, hostBins, NUM_BINS);

  free(hostBins);
  free(hostInput);
  return 0;
}
