#include "hip/hip_runtime.h"
#include <wb.h>
#include <cmath>
#include <stdio.h>
#include <string.h>
#include "hip/hip_runtime.h"
#include ""

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      return -1;                                                          \
    }                                                                     \
  } while (0)

#define Mask_width 5
#define Mask_radius Mask_width / 2
#define TILE_WIDTH 16
#define w (TILE_WIDTH + Mask_width - 1)
#define clamp(x) (min(max((x), 0.0), 1.0))

//@@ INSERT CODE HERE

__global__ void convolution(float *deviceInputImageData, const float * __restrict__ deviceMaskData,
	float *deviceOutputImageData, int imageChannels,
	int imageWidth, int imageHeight)
{
	unsigned int row = threadIdx.y + blockDim.y * blockIdx.y;
	unsigned int col = threadIdx.x + blockDim.x * blockIdx.x;

	__shared__ float s_input[w]; 

	if (col < imageHeight && row < imageWidth)
	{
		int pixelVal = 0;

		int colStart = col - Mask_radius;
		int rowStart = row - Mask_radius;

		for (int j = 0; j < Mask_width; j++)
		{
			for (int i = 0; i < Mask_width; i++)
			{
				int curRow = rowStart + j;
				int curCol = colStart + i;	

				if (curRow > -1 && curCol > -1 && curRow < imageHeight && curCol < imageWidth)
				{
					pixelVal += deviceInputImageData[curRow * w + curCol] * deviceMaskData[j * Mask_width + i];
				}
			}
		}

		deviceOutputImageData[row * w + col] = (unsigned char)pixelVal;
	}
}

int main(int argc, char *argv[]) {
  wbArg_t arg;
  int maskRows;
  int maskColumns;
  int imageChannels;
  int imageWidth;
  int imageHeight;
  char *inputImageFile;
  char *inputMaskFile;
  wbImage_t inputImage;
  wbImage_t outputImage;
  float *hostInputImageData;
  float *hostOutputImageData;
  float *hostMaskData;
  float *deviceInputImageData;
  float *deviceOutputImageData;
  float *deviceMaskData;

  arg = wbArg_read(argc, argv); /* parse the input arguments */

  inputImageFile = wbArg_getInputFile(arg, 0);
  inputMaskFile  = wbArg_getInputFile(arg, 1);

  inputImage   = wbImport(inputImageFile);
  hostMaskData = (float *)wbImport(inputMaskFile, &maskRows, &maskColumns);

  assert(maskRows == 5);    /* mask height is fixed to 5 in this mp */
  assert(maskColumns == 5); /* mask width is fixed to 5 in this mp */

  imageWidth    = wbImage_getWidth(inputImage);
  imageHeight   = wbImage_getHeight(inputImage);
  imageChannels = wbImage_getChannels(inputImage);

  outputImage = wbImage_new(imageWidth, imageHeight, imageChannels);

  hostInputImageData  = wbImage_getData(inputImage);
  hostOutputImageData = wbImage_getData(outputImage);

  wbTime_start(GPU, "Doing GPU Computation (memory + compute)");

  wbTime_start(GPU, "Doing GPU memory allocation");
  //@@ INSERT CODE HERE

  hipMalloc((void**)&deviceInputImageData, imageHeight * imageWidth * imageChannels * sizeof(char));
  hipMalloc((void**)&deviceOutputImageData, imageHeight * imageWidth * imageChannels * sizeof(char));
  hipMalloc((void**)&deviceMaskData, Mask_width * Mask_width * sizeof(int));
  
  wbTime_stop(GPU, "Doing GPU memory allocation");

  wbTime_start(Copy, "Copying data to the GPU");
  //@@ INSERT CODE HERE

  hipMemcpy(deviceInputImageData, hostInputImageData, imageHeight * imageHeight * imageChannels * sizeof(char), hipMemcpyHostToDevice);
  hipMemcpy(deviceOutputImageData, hostOutputImageData, imageHeight * imageHeight * imageChannels * sizeof(char), hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(deviceMaskData), hostMaskData, Mask_width * Mask_width * sizeof(int));
  wbTime_stop(Copy, "Copying data to the GPU");

  wbTime_start(Compute, "Doing the computation on the GPU");
  //@@ INSERT CODE HERE
  dim3 dimGrid(TILE_WIDTH, TILE_WIDTH);
  dim3 dimBlock(ceil((float)imageHeight/TILE_WIDTH), ceil((float)imageWidth/TILE_WIDTH));

  convolution<<<dimGrid, dimBlock>>>(deviceInputImageData, deviceMaskData,
                                     deviceOutputImageData, imageChannels,
                                     imageWidth, imageHeight);
  wbTime_stop(Compute, "Doing the computation on the GPU");

  wbTime_start(Copy, "Copying data from the GPU");
  //@@ INSERT CODE HERE
  hipMemcpy(hostOutputImageData, deviceOutputImageData,
             imageWidth * imageHeight * imageChannels * sizeof(float),
             hipMemcpyDeviceToHost);

 printf("Data from Host %f" , hostOutputImageData[0]);
 printf("Data from Host %f" , hostOutputImageData[1]);
 printf("Data from Host %f" , hostOutputImageData[2]);
  wbTime_stop(Copy, "Copying data from the GPU");

  wbTime_stop(GPU, "Doing GPU Computation (memory + compute)");

  wbSolution(arg, outputImage);

  //@@ Insert code here

  hipFree(deviceInputImageData);
  hipFree(deviceOutputImageData);
  hipFree(deviceMaskData);

  free(hostMaskData);
  wbImage_delete(outputImage);
  wbImage_delete(inputImage);

  return 0;
}
