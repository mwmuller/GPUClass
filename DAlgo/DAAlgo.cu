#include "hip/hip_runtime.h"
﻿#include <sstream>
#include <vector>
#include <iostream>
#include <float.h>
#include <cmath>
#include <stdio.h>
#include <string.h>
#include "hip/hip_runtime.h"
#include ""

#define NUM_ASYNCHRONOUS_ITERATIONS 20  // Number of async loop iterations before attempting to read results back

#define BLOCK_SIZE 16

#define floatScalar 10000


// --- The graph data structure is an adjacency list.
typedef struct {

	// --- Contains the integer offset to point to the edge list for each vertex
	int *vertexArray;

	// --- Overall number of vertices
	int numVertices;

	// --- Contains the "destination" vertices each edge is attached to
	int *edgeArray;

	// --- Overall number of edges
	int numEdges;

	// --- Contains the weight of each edge
	float *weightArray;

} GraphData;

/**********************************/
/* GENERATE RANDOM GRAPH FUNCTION */
/**********************************/
void generateRandomGraph(GraphData *graph, int numVertices, int neighborsPerVertex) {

	graph->numVertices = numVertices;
	graph->vertexArray = (int *)malloc(graph->numVertices * sizeof(int));
	graph->numEdges = numVertices * neighborsPerVertex;
	graph->edgeArray = (int *)malloc(graph->numEdges * sizeof(int));
	graph->weightArray = (float *)malloc(graph->numEdges * sizeof(float));

	for (int i = 0; i < graph->numVertices; i++) graph->vertexArray[i] = i * neighborsPerVertex;

	int *tempArray = (int *)malloc(neighborsPerVertex * sizeof(int));
	for (int k = 0; k < numVertices; k++) {
		for (int l = 0; l < neighborsPerVertex; l++) tempArray[l] = INT_MAX;
		for (int l = 0; l < neighborsPerVertex; l++) {
			bool goOn = false;
			int temp;
			while (goOn == false) {
				goOn = true;
				temp = (rand() % graph->numVertices); // move to 0;
				for (int t = 0; t < neighborsPerVertex; t++)
					if (temp == tempArray[t]) goOn = false;
				if (temp == k) goOn = false;
				if (goOn == true) tempArray[l] = temp;
			}
			graph->edgeArray[k * neighborsPerVertex + l] = temp;
			graph->weightArray[k * neighborsPerVertex + l] = (float)(rand() % 1000) / 1000.0f;
		}
	}
}

/************************/
/* minDistance FUNCTION */
/************************/
// --- Finds the vertex with minimum distance value, from the set of vertices not yet included in shortest path tree
int minDistance(float *shortestDistances, bool *finalizedVertices, const int sourceVertex, const int N) {

	// --- Initialize minimum value
	int minIndex = sourceVertex;
	float min = FLT_MAX;

	for (int v = 0; v < N; v++)
		if (finalizedVertices[v] == false && shortestDistances[v] <= min) min = shortestDistances[v], minIndex = v;

	return minIndex;
}

/************************/
/* dijkstraCPU FUNCTION */ // This will remain unchanged
/************************/
void dijkstraCPU(float *graph, float *h_shortestDistances, int sourceVertex, const int N) {

	// --- h_finalizedVertices[i] is true if vertex i is included in the shortest path tree
	//     or the shortest distance from the source node to i is finalized
	bool *h_finalizedVertices = (bool *)malloc(N * sizeof(bool));

	// --- Initialize h_shortestDistancesances as infinite and h_shortestDistances as false
	for (int i = 0; i < N; i++) h_shortestDistances[i] = FLT_MAX, h_finalizedVertices[i] = false;

	// --- h_shortestDistancesance of the source vertex from itself is always 0
	h_shortestDistances[sourceVertex] = 0.f;

	// --- Dijkstra iterations
	for (int iterCount = 0; iterCount < N - 1; iterCount++) {

		// --- Selecting the minimum distance vertex from the set of vertices not yet
		//     processed. currentVertex is always equal to sourceVertex in the first iteration.
		int currentVertex = minDistance(h_shortestDistances, h_finalizedVertices, sourceVertex, N);

		// --- Mark the current vertex as processed
		h_finalizedVertices[currentVertex] = true;

		// --- Relaxation loop
		for (int v = 0; v < N; v++) {

			// --- Update dist[v] only if it is not in h_finalizedVertices, there is an edge
			//     from u to v, and the cost of the path from the source vertex to v through
			//     currentVertex is smaller than the current value of h_shortestDistances[v]
			if (!h_finalizedVertices[v] &&
				graph[currentVertex * N + v] &&
				h_shortestDistances[currentVertex] != FLT_MAX &&
				h_shortestDistances[currentVertex] + graph[currentVertex * N + v] < h_shortestDistances[v])

				h_shortestDistances[v] = h_shortestDistances[currentVertex] + graph[currentVertex * N + v];
		}
	}
}

/***************************/
/* MASKARRAYEMPTY FUNCTION */
/***************************/
// --- Check whether all the vertices have been finalized. This tells the algorithm whether it needs to continue running or not.
bool allFinalizedVertices(bool *finalizedVertices, int numVertices) {

	for (int i = 1; i < numVertices; i++)
	{
		if (finalizedVertices[i] == false)
		{
			printf("Index that is true: %d", i);
			return false;
		}
	}


	return true;
}

/*************************/
/* ARRAY INITIALIZATIONS */
/*************************/
__global__ void initializeArrays(bool * __restrict__ d_finalizedVertices, float* __restrict__ d_shortestDistances, float* __restrict__ d_updatingShortestDistances,
	const int sourceVertex, const int numVertices) {

	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	if (tid < numVertices) {

		d_finalizedVertices[tid] = false;
		d_shortestDistances[tid] = FLT_MAX;
		d_updatingShortestDistances[tid] = FLT_MAX;

		/*
			if (sourceVertex == tid) {

				d_finalizedVertices[tid] = true;
				d_shortestDistances[tid] = 0.f;
				d_updatingShortestDistances[tid] = 0.f;
			}

			else {

				d_finalizedVertices[tid] = false;
				d_shortestDistances[tid] = FLT_MAX;
				d_updatingShortestDistances[tid] = FLT_MAX;
			}
			*/
	}
}

/**************************/
/* DIJKSTRA GPU KERNEL #1 */
/**************************/
__global__  void Kernel1(const int * __restrict__ vertexArray, const int* __restrict__ edgeArray,
	const float * __restrict__ weightArray, bool * __restrict__ finalizedVertices, float * __restrict__ shortestDistances,
	float * __restrict__ updatingShortestDistances, const int numVertices, const int numEdges, int * tempIntMinArray) {

	int tid = blockIdx.x*blockDim.x + threadIdx.x;

	if (tid < numVertices) {

		if (finalizedVertices[tid] != true) {

			int edgeStart = tid * (numEdges / numVertices), edgeEnd; // get the edge index that we start at

			// Check if we are beyond the number of verticies that we can check
			if (tid + 1 < (numVertices)) edgeEnd = (tid + 1) * (numEdges / numVertices); // Check if we are in bounds. 
			else                         edgeEnd = numEdges; // We are at the max.

			for (int edge = edgeStart; edge < edgeEnd; edge++) {
				int nid = edgeArray[edge]; // get the ID which will be associated with a vertex
				tempIntMinArray[nid] = updatingShortestDistances[nid] * floatScalar; // Assign the current value into the temp array
				int shortWeightMin;
				if (shortestDistances[nid] == FLT_MAX) // We need to check if the edge has been processed
				{
					shortestDistances[nid] = 0;
				}
				
				shortWeightMin = ((shortestDistances[nid] * floatScalar) + (weightArray[edge] * floatScalar));

				atomicMin(&tempIntMinArray[nid], shortWeightMin); // assigns minimum value to uSD pointer

				__syncthreads(); // Sync before assigning updatingShortestInt version to float array.
				if (tempIntMinArray[nid] != 0 && tid < nid)
				{
					shortestDistances[nid] = shortWeightMin;
					updatingShortestDistances[nid] = (float)(tempIntMinArray[nid]) / floatScalar;
				}
			}
		}
	}
}

/**************************/
/* DIJKSTRA GPU KERNEL #2 */
/**************************/
__global__  void Kernel2(const int * __restrict__ vertexArray, const int * __restrict__ edgeArray, const float* __restrict__ weightArray,
	bool * __restrict__ finalizedVertices, float* __restrict__ shortestDistances, float* __restrict__ updatingShortestDistances,
	const int numVertices) {

	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	if (tid < numVertices) {

		if (shortestDistances[tid] > updatingShortestDistances[tid]) {
			shortestDistances[tid] = updatingShortestDistances[tid];
			finalizedVertices[tid] = true;
		}

		updatingShortestDistances[tid] = shortestDistances[tid];
	}
}

/************************/
/* dijkstraGPU FUNCTION */
/************************/
void dijkstraGPU(GraphData *graph, const int sourceVertex, float * __restrict__ h_shortestDistances) {

	// --- Create device-side adjacency-list, namely, vertex array Va, edge array Ea and weight array Wa from G(V,E,W)
	int     *d_vertexArray;         hipMalloc(&d_vertexArray, sizeof(int)   * graph->numVertices);
	int     *d_edgeArray;           hipMalloc(&d_edgeArray, sizeof(int)   * graph->numEdges);
	float   *d_weightArray;         hipMalloc(&d_weightArray, sizeof(float) * graph->numEdges);

	// --- Copy adjacency-list to the device
	hipMemcpy(d_vertexArray, graph->vertexArray, sizeof(int)   * graph->numVertices, hipMemcpyHostToDevice);
	hipMemcpy(d_edgeArray, graph->edgeArray, sizeof(int)   * graph->numEdges, hipMemcpyHostToDevice);
	hipMemcpy(d_weightArray, graph->weightArray, sizeof(float) * graph->numEdges, hipMemcpyHostToDevice);

	// --- Create mask array Ma, cost array Ca and updating cost array Ua of size V
	bool    *d_finalizedVertices;           hipMalloc(&d_finalizedVertices, sizeof(bool)   * graph->numVertices);
	float   *d_shortestDistances;           hipMalloc(&d_shortestDistances, sizeof(float) * graph->numVertices);
	float   *d_updatingShortestDistances;   hipMalloc(&d_updatingShortestDistances, sizeof(float) * graph->numVertices);
	int		*d_tempIntMinArray;				hipMalloc(&d_tempIntMinArray, sizeof(int) * graph->numVertices);
	bool *h_finalizedVertices = (bool *)malloc(sizeof(bool) * graph->numVertices);

	// --- Initialize mask Ma to false, cost array Ca and Updating cost array Ua to \u221e
	initializeArrays << <ceil((float)(graph->numVertices)/BLOCK_SIZE), BLOCK_SIZE >> > (d_finalizedVertices, d_shortestDistances,
		d_updatingShortestDistances, sourceVertex, graph->numVertices);
	//hipPeekAtLastError());
	hipDeviceSynchronize();

	// --- Read mask array from device -> host
	hipMemcpy(h_finalizedVertices, d_finalizedVertices, sizeof(bool) * graph->numVertices, hipMemcpyDeviceToHost);

	//while (!allFinalizedVertices(h_finalizedVertices, graph->numVertices)) {

		// --- In order to improve performance, we run some number of iterations without reading the results.  This might result
		//     in running more iterations than necessary at times, but it will in most cases be faster because we are doing less
		//     stalling of the GPU waiting for results.
		for (int asyncIter = 0; asyncIter < NUM_ASYNCHRONOUS_ITERATIONS; asyncIter++) {

			Kernel1 << <(ceil((float)(graph->numVertices)/BLOCK_SIZE)), BLOCK_SIZE >> > (d_vertexArray, d_edgeArray, d_weightArray, d_finalizedVertices, d_shortestDistances,
				d_updatingShortestDistances, graph->numVertices, graph->numEdges, d_tempIntMinArray);

			hipDeviceSynchronize();
			Kernel2 << <(ceil((float)(graph->numVertices) / BLOCK_SIZE)), BLOCK_SIZE >> > (d_vertexArray, d_edgeArray, d_weightArray, d_finalizedVertices, d_shortestDistances, d_updatingShortestDistances,
				graph->numVertices);
			hipDeviceSynchronize();
		}

		hipMemcpy(h_finalizedVertices, d_finalizedVertices, sizeof(bool) * graph->numVertices, hipMemcpyDeviceToHost);
	//}

	// --- Copy the result to host
	hipMemcpy(h_shortestDistances, d_shortestDistances, sizeof(float) * graph->numVertices, hipMemcpyDeviceToHost);

	free(h_finalizedVertices);

	hipFree(d_vertexArray);
	hipFree(d_edgeArray);
	hipFree(d_weightArray);
	hipFree(d_finalizedVertices);
	hipFree(d_shortestDistances);
	hipFree(d_updatingShortestDistances);
}

/****************/
/* MAIN PROGRAM */
/****************/
int main() {

	// --- Number of graph vertices
	int numVertices = 8;

	// --- Number of edges per graph vertex
	int neighborsPerVertex = 6;

	// --- Source vertex
	int sourceVertex = 0;

	// --- Allocate memory for arrays
	GraphData graph;
	generateRandomGraph(&graph, numVertices, neighborsPerVertex);

	// --- From adjacency list to adjacency matrix.
	//     Initializing the adjacency matrix
	float *weightMatrix = (float *)malloc(numVertices * numVertices * sizeof(float));
	for (int k = 0; k < numVertices * numVertices; k++) weightMatrix[k] = FLT_MAX;

	// --- Displaying the adjacency list and constructing the adjacency matrix
	printf("Adjacency list\n");
	for (int k = 0; k < numVertices; k++) weightMatrix[k * numVertices + k] = 0.f;
	for (int k = 0; k < numVertices; k++)
		for (int l = 0; l < neighborsPerVertex; l++) {
			weightMatrix[k * numVertices + graph.edgeArray[graph.vertexArray[k] + l]] = graph.weightArray[graph.vertexArray[k] + l];
			printf("Vertex nr. %i; Edge nr. %i; Weight = %f\n", k, graph.edgeArray[graph.vertexArray[k] + l],
				graph.weightArray[graph.vertexArray[k] + l]);
		}

	for (int k = 0; k < numVertices * neighborsPerVertex; k++)
		printf("%i %i %f\n", k, graph.edgeArray[k], graph.weightArray[k]);
	
	// --- Displaying the adjacency matrix
	if (numVertices < 100)
	{
		printf("\nAdjacency matrix\n");
		for (int k = 0; k < numVertices; k++) {
			for (int l = 0; l < numVertices; l++)
				if (weightMatrix[k * numVertices + l] < FLT_MAX)
					printf("%1.3f\t", weightMatrix[k * numVertices + l]);
				else
					printf("--\t");
			printf("\n");
		}
	}
	else
	{
		// do nothing because we don't have that kind of time
	}
	// --- Running Dijkstra on the CPU
	float *h_shortestDistancesCPU = (float *)malloc(numVertices * sizeof(float));
	dijkstraCPU(weightMatrix, h_shortestDistancesCPU, sourceVertex, numVertices);

	printf("\nCPU results\n");
	for (int k = 0; k < numVertices; k++) printf("From vertex %i to vertex %i = %f\n", sourceVertex, k, h_shortestDistancesCPU[k]);

	// --- Allocate space for the h_shortestDistancesGPU
	float *h_shortestDistancesGPU = (float*)malloc(sizeof(float) * graph.numVertices);
	dijkstraGPU(&graph, sourceVertex, h_shortestDistancesGPU);

	printf("\nGPU results\n");
	for (int k = 0; k < numVertices; k++) printf("From vertex %i to vertex %i = %f\n", sourceVertex, k, h_shortestDistancesGPU[k]);

	free(h_shortestDistancesCPU);
	free(h_shortestDistancesGPU);

	return 0;
}