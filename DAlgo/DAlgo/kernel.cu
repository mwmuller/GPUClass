#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <string>
#include <stdlib.h>
#include <cmath>
using namespace std;

// Number of vertices in the graph
#define V 9
void convertTo1DArray();
// A utility function to find the vertex with minimum distance value, from
// the set of vertices not yet included in shortest path tree
void minDistance(int dist[], bool sptSet[], int* &min_index)
{

	// Initialize min value
	int min = INT_MAX;

	for (int v = 0; v < V; v++)
		if (sptSet[v] == false && dist[v] <= min)
			min = dist[v], min_index = new int(v);
}

__global__ void print2dArr(int *arr)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	printf("we are here");
	for (int x = 0; x < V; x++)
	{
		printf("%d", arr[i*V+x]);

		if (x + 1 % V == 0) printf("\n");
	}
}
__global__ void calcShortest(int *graph, int* dist, bool* sptSet, int minIndex)
{
	// We need to have each thread check a different dist location. If they all check the name location, then
	// We will have the highest thread < V update the value in dist arr
	int thread = blockIdx.x*blockDim.x + threadIdx.x;
	if (thread < V)
	{
	//	if (!sptSet[thread] && graph[minIndex][thread] && dist[minIndex] != INT_MAX
			//&& ((dist[minIndex] + graph[minIndex][thread]) < dist[thread]))
			//dist[thread] = dist[thread] + graph[minIndex][thread];
	}
}

// A utility function to print the constructed distance array
__global__ void printSolution(int* dist)
{
	printf("Vertex \t Distance from Source\n");
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	printf(" \t\t%d\n", dist[i]);
}

// Function that implements Dijkstra's single source shortest path algorithm
// for a graph represented using adjacency matrix representation
int* dijkstra(int *graph, int src)
{
	hipError_t hipError_t;

	size_t dSize = V * V * sizeof(int);
	// init an array of ptrs to more arrays
	int test2dArr[sizeof(dSize)];
	int *dev_graph;
	int* dist = new int[V]; // The output array.  dist[i] will hold the shortest
	int* dev_Dist = { 0 };

	bool* sptSet = new bool[V]; // sptSet[i] will be true if vertex i is included in shortest

	// setting the max distance
	for (int i = 0; i < V; i++)
		dist[i] = INT_MAX, sptSet[i] = false;

	dim3 threads = 32; // declaring the amount of threads based on the size of V (nodes per dimension)
	dim3 blocks = ((ceil(static_cast<float>(V) / static_cast<float>(32)))); // defining the number of blocks that will be required.
	// Initialize all distances as INFINITE and stpSet[] as false

	// Distance of source vertex from itself is always 0
	dist[src] = 0;

	// Find shortest path for all vertices
	for (int count = 0; count < V - 1; count++) {
		int *u = 0;
		int minIndex = 0;
		// Pick the minimum distance vertex from the set of vertices not
		// yet processed. u is always equal to src in the first iteration.
		minDistance(dist, sptSet, u);
		minIndex = *u;
		// Mark the picked vertex as processed
		sptSet[minIndex] = true;

		// Alloc and then copy ptr to 
		hipError_t = hipMalloc((void**)&dev_Dist, V * sizeof(int));
		if (hipError_t != hipSuccess)
		{
			fprintf(stderr, "%s", hipError_t);
		}
		hipError_t = hipMemcpy(dev_Dist, dist, V * sizeof(int), hipMemcpyHostToDevice);
		if (hipError_t != hipSuccess)
		{
			fprintf(stderr, "%s", hipError_t);
		}
		hipMalloc((void**)&dev_graph, dSize);
		hipMemcpy2D(dev_graph, dSize, graph, V, V, V, hipMemcpyHostToDevice);

		hipMemcpy(test2dArr, dev_graph, dSize, hipMemcpyDeviceToHost);
		
		calcShortest<<<blocks, threads >>>(dev_graph, dev_Dist, sptSet, minIndex);

		hipDeviceSynchronize();
		hipMemcpy(dist, dev_Dist, dSize, hipMemcpyDeviceToHost);
		// Update dist[v] only if is not in sptSet, there is an edge from
		// u to v, and total weight of path from src to  v through u is
		// smaller than current value of dist[v]

	}



	return dist;
}

// driver program to test above function
int main()
{
	size_t dSize = V * V * sizeof(int);
	int* hostDist = (int*)malloc(dSize);

	/* Let us create the example graph discussed above */
	int graph[V][V] = { { 0, 4, 0, 0, 0, 0, 0, 8, 0 }, // This might need to be converted to a 1d Matrix for simplicity
						{ 4, 0, 8, 0, 0, 0, 0, 11, 0 },
						{ 0, 8, 0, 7, 0, 4, 0, 0, 2 },
						{ 0, 0, 7, 0, 9, 14, 0, 0, 0 },
						{ 0, 0, 0, 9, 0, 10, 0, 0, 0 },
						{ 0, 0, 4, 14, 10, 0, 2, 0, 0 },
						{ 0, 0, 0, 0, 0, 2, 0, 1, 6 },
						{ 8, 11, 0, 0, 0, 0, 1, 0, 7 },
						{ 0, 0, 2, 0, 0, 0, 6, 7, 0 } };

	// We can convert the 2d array into a 1d array in qhich we construct a map. 
	// We can create an array of size ArrSize ==> [(1+n)n]/2 n is the width and 
	// // ceil(sqrt(ArrSize*2)) will provide us the width/height of our 2d array.
	// this will produce an array of size ceil(sqrt(n*2))^2 * sizeof(int)

	// ceil(sqrt(n*2)) will provide us the width/height of our 2d array.

	convertTo1DArray();
	memcpy(hostDist, graph, dSize);
	hipMemcpy2D(hostDist, dSize, graph, dSize, V, V, hipMemcpyHostToHost);
	int* sol = dijkstra(hostDist, 0);

	printSolution << <1, V >> > (sol);
	hipDeviceSynchronize();

	return 0;
}

int getMallocSize(int arrayPitch)
{
	int size = pow(arrayPitch, 2.0) * sizeof(int);
	return size;
}
void convertTo1DArray()
{
	int testArr[6] = { 1, 0, 2, 5, 4, 6 };
	int elements = sizeof(testArr) / sizeof(int);
	int arrPitch = ceil(sqrt(static_cast<double>(elements * 2)));
	int totalSize = getMallocSize(arrPitch);
	int *output = (int*)malloc(getMallocSize(arrPitch));
	memset(output, 0, totalSize);
	for (int i = 0; i < arrPitch; i++)
	{
		float diagonal = floor(static_cast<float>(i)/ static_cast<float>((arrPitch + 1)));

		for (int x = 0; x < arrPitch - 1; x++)
		{
			output[(x * 5) + (1 + i)] = testArr[(0 + x + i) % elements]; // currently working on a solution to this
		}
	}
}