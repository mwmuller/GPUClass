#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <string>
#include <stdlib.h>
#include <cmath>
#include <iostream>
using namespace std;


// A utility function to find the vertex with minimum distance value, from
// the set of vertices not yet included in shortest path tree
void minDistance(int* dist, bool* sptSet, int *min_index, int arrPitch)
{

	// Initialize min value
	int min = INT_MAX;

	for (int v = 0; v < arrPitch; v++)
		if (sptSet[v] == false && dist[v] <= min)
			min = dist[v], *min_index = v;
}

__global__ void calcShortest(int *graph, int* dist, bool* sptSet, int minIndex, int arrPitch)
{
	// We need to have each thread check a different dist location. If they all check the name location, then
	// We will have the highest thread < V update the value in dist arr

	int thread = blockIdx.x*blockDim.x + threadIdx.x;
	if (thread < arrPitch)
	{
		int graphRow = thread * arrPitch;
		for (int i = 0; i < arrPitch; i++)
		{
			if (!sptSet[thread] && graph[graphRow + i] && dist[thread] != INT_MAX
				&& ((dist[thread] + graph[graphRow + i]) < dist[thread]))
				dist[thread] = dist[thread] + graph[graphRow + i];
		}
	}
}

// A utility function to print the constructed distance array
void printSolution(int* dist, int arrPitch)
{
	for(int i = 0; i < arrPitch - 1; i++)
	{
		printf("Vertex \t Distance from Source\n");
		printf(" \t\t%d\n", dist[i]);
	}
}

// Function that implements Dijkstra's single source shortest path algorithm
// for a graph represented using adjacency matrix representation
void dijkstra(int *graph, int src, int* arrPitch)
{
	hipError_t hipError_t;
	int thisPitch = *arrPitch;
	size_t dSize = thisPitch * thisPitch * sizeof(int);
	// init an array of ptrs to more arrays for device
	int *dev_graph = 0;
	int* dist = new int[thisPitch]; // The output array.  dist[i] will hold the shortest
	int* dev_Dist = 0;
	bool* devicesptSet = false;
	bool* sptSet = new bool[thisPitch]; // sptSet[i] will be true if vertex i is included in shortest

	// setting the max distance
	for (int i = 0; i < thisPitch; i++)
		dist[i] = INT_MAX, sptSet[i] = false;

	dim3 threads = 32; // declaring the amount of threads based on the size of V (nodes per dimension)
	dim3 blocks = ((ceil(static_cast<float>(thisPitch) / static_cast<float>(32)))); // defining the number of blocks that will be required.
	// Initialize all distances as INFINITE and stpSet[] as false

	// Distance of source vertex from itself is always 0
	dist[src] = 0;
	hipMalloc((void**)&devicesptSet, thisPitch*sizeof(bool));
	hipMalloc((void**)&dev_graph, dSize);
	hipMemcpy(dev_graph, graph, dSize, hipMemcpyHostToDevice);
		// Find shortest path for all vertices
		int *u = new int(0);
		int minIndex = 0;
		// Pick the minimum distance vertex from the set of vertices not
		// yet processed. u is always equal to src in the first iteration.
		minDistance(dist, sptSet, u, thisPitch);
		minIndex = *u;
		// Mark the picked vertex as processed
		sptSet[minIndex] = true;

		hipMemcpy(devicesptSet, sptSet, dSize, hipMemcpyHostToDevice);
		hipMalloc((void**)&dev_graph, dSize);
		hipMemcpy(dev_graph, graph, dSize, hipMemcpyHostToDevice);
		// Alloc and then copy ptr to 
		hipError_t = hipMalloc((void**)&dev_Dist, thisPitch * sizeof(int));
		if (hipError_t != hipSuccess)
		{
			fprintf(stderr, "%s", hipError_t);
		}
		hipError_t = hipMemcpy(dev_Dist, dist, thisPitch * sizeof(int), hipMemcpyHostToDevice);
		if (hipError_t != hipSuccess)
		{
			fprintf(stderr, "%s", hipError_t);
		}

		calcShortest << <blocks, threads >> > (dev_graph, dev_Dist, sptSet, minIndex, thisPitch);
		hipMemcpy(dist, dev_Dist, thisPitch * sizeof(int), hipMemcpyDeviceToHost);

		hipFree(dev_graph);
		hipFree(dev_Dist);

		printSolution(dist, thisPitch);
		// Update dist[v] only if is not in sptSet, there is an edge from
		// u to v, and total weight of path from src to  v through u is
		// smaller than current value of dist[v]
}

__global__ void create1DMapUnstructured(int* inputArr, int *outputArr, int *inPitch)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int arrPitch = *inPitch;
	int inputElements = (arrPitch*(arrPitch - 1)) / 2;
	int topArr = inputElements-((((i + 2) * (i + 1)) / 2));
	int botArr = topArr;

	if (i < arrPitch - 1)
	{
		for (int x = 0; x < (arrPitch - (arrPitch - (1 + i))); x++)
		{
			int getit = inputArr[topArr];
			int index = (x + 1) + (arrPitch + 1)*(arrPitch - 2 - i);
			outputArr[index] = inputArr[topArr];
			topArr++;
		}
		// 0, y > 3 y++
		for (int y = arrPitch - (arrPitch - (1 + i)); y > 0; y--)
		{
			int getity = inputArr[botArr];
			int indexy = (arrPitch*(arrPitch - 1 - (y-1))) + (arrPitch - 2 - i);
			outputArr[indexy] = inputArr[botArr];
			botArr++;
		}
	}
}

// driver program to test above function
int main()
{
	int* hostGraph;
	int* arrPitch = new int(0);

	// Device values needed
	int* devInGraph;
	int* devOutGraph = 0;
	int* devDist;
	int* devInArrSize;
	int* devArrPitch;
	/* Let us create the example graph discussed above */
	/*
	int graph[] = { 4, 1, 0, 5, 0, 0, 8, 0,
					8, 0, 0, 0, 0, 11, 0,
					7, 0, 4, 0, 2, 2,
					9, 14, 0, 0, 0,
					10, 0, 0, 0,
					2, 0, 0,
					1, 6,
					7 }; // this is out simplified graph
			*/	
	int graph[] = { 1,0,2,5,4,6,0,1,0,3};

	// We can create an array of size ArrSize ==> [(1+n)n]/2 n is the width and 
	// ceil(sqrt(ArrSize*2)) will provide us the width/height of our 2d array.
	// this will produce an array of size ceil(sqrt(n*2))^2 * sizeof(int)
	int elements = sizeof(graph) / sizeof(int);

	// ceil(sqrt(ArrSize*2)) will provide us the width/height of our 2d array.
	*arrPitch = ceil(sqrt(static_cast<double>(elements * 2)));

	// Declare the max cuda size needed
	int totalCudaMallocSize = (*arrPitch) * (*arrPitch) * sizeof(int);


	// ceil(sqrt(n*2)) will provide us the width/height of our 2d array.
	dim3 threads = 32; // declaring the amount of threads based on the size of V (nodes per dimension)
	dim3 blocks = ((ceil(static_cast<float>(*arrPitch) / static_cast<float>(32)))); // defining the number of blocks that will be required.

	hostGraph = (int*)malloc(totalCudaMallocSize);
	memset(hostGraph, 0, totalCudaMallocSize);

	hipMalloc((void**)&devInGraph, elements * sizeof(int));
	hipMemcpy(devInGraph, graph, elements * sizeof(int), hipMemcpyHostToDevice);


	hipMalloc((void**)&devOutGraph, totalCudaMallocSize);
	hipMemset(devOutGraph, 0, totalCudaMallocSize);

	hipMalloc((void**)&devArrPitch, sizeof(int));
	hipMemcpy(devArrPitch, arrPitch, sizeof(int), hipMemcpyHostToDevice);
	
	hipMalloc((void**)&devInArrSize, sizeof(int));
	hipMemcpy(devArrPitch, arrPitch, sizeof(int), hipMemcpyHostToDevice);

	create1DMapUnstructured<<<blocks, threads>>>(devInGraph, devOutGraph, devArrPitch);

	hipDeviceSynchronize();
	int hostSize = (*arrPitch)* (*arrPitch);
	hipMemcpy(hostGraph, devOutGraph, (*arrPitch)* (*arrPitch) * sizeof(int), hipMemcpyDeviceToHost);

	for (int i = 0; i < hostSize; i++)
	{
		int getit = hostGraph[i];
		if (i % *arrPitch == 0)
		{
			printf("\n");
		}
		printf("%d  ", getit);
	}
	dijkstra(hostGraph, 0, arrPitch);
	
	hipDeviceSynchronize();

	return 0;
}
