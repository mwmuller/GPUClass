﻿
#include "hip/hip_runtime.h"


#include <stdio.h>

#define threadSIZE  1024 // max thread count per cuda
#define blockSize   16 // max number of blocks
#define arrScale 75 // amount of times to multiple threads
#define arrSize  (threadSIZE * arrScale)

hipError_t addWithCuda(int *c, int *a, int *b, unsigned int size);

__global__ void addKernel(int *c, int *a, int *b)
{
    int j = threadIdx.x;
    c[j] = a[j] + b[j];

    for (int i = 1; i < arrScale; i++)
    {
        int index = (threadSIZE * i) + j;

        c[index] = a[index] + b[index];
    }
}

__global__ void printArr(int* arr)
{
    for (int i = 0; i < arrSize; i++)
    {
        printf("%d, ", arr[i]);
    }
}

__global__ void fillArrs(int* aArray, int *bArray)
{
    int j = threadIdx.x;
    
    aArray[j] = j + 1;
    bArray[j] = j * 2;
    // move backwards from the max

    for (int i = 1; i < arrScale; i++)
    {
        int index = (threadSIZE * i) + j;
        aArray[index] = index + 1; // 1024 * the scalar + the initial index
        bArray[index] = index * 2; // 1024 * the scalar + the initial index
    }

}

int main()
{

    hipError_t cudaStatus;
    // an array of 4 points;
    int a[arrSize] = { 0 };
    int b[arrSize] = { 0 };
    int c[arrSize] = { 0 };

    size_t sizeCount = arrSize * sizeof(int);

    int* dev_a = new int[arrSize];
    int* dev_b = new int[arrSize];
    int* dev_c = new int[arrSize];

    cudaStatus = hipMalloc((void**)&dev_a, sizeCount);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy awdawd failed!");
        return 1;
    }
    cudaStatus = hipMalloc((void**)&dev_b, sizeCount);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy awdawd failed!");
        return 1;
    }
    cudaStatus = hipMemcpy(dev_a, a, sizeCount, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy awdawd failed!");
        return 1;
    }
    cudaStatus = hipMemcpy(dev_b, b, sizeCount, hipMemcpyHostToDevice);

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy awdawd failed!");
        return 1;
    }

    // filling array with numbers
    fillArrs <<<1, threadSIZE>>>(dev_a, dev_b);

    cudaStatus = hipMemcpy(a, dev_a, arrSize * sizeof(int), hipMemcpyDeviceToHost);
    cudaStatus = hipMemcpy(b, dev_b, arrSize * sizeof(int), hipMemcpyDeviceToHost);
    // Add vectors in parallel.

    cudaStatus = addWithCuda(c, a, b, arrSize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    cudaStatus = hipMalloc((void**)&dev_c, arrSize * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        return 1;
    }

    cudaStatus = hipMemcpy(dev_c, c, arrSize * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        return 1;
    }
    
    printf("Array C {");
    printArr<<<1, 1>>>(dev_c);
    printf("} \n");
    

    printf("done");

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, int *a, int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output).
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, threadSIZE>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
